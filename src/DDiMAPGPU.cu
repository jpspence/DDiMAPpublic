#include "hip/hip_runtime.h"
//============================================================================
// Name        : DDiMAP.cpp
// Author      : Androwis Abumoussa
// Version     :
// Copyright   : All Rights Reserved
// Description : DDiMAP in C
//============================================================================
#include "DDiMAPGPU.h"
#include <getopt.h>
#include <time.h>
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <string>

// Default file.
// char *file = "data/Burack_128F/128F_Gen1_Frag_WithBcl2Sanger_sorted.bam";
string file  = "data/128test_Gen1_example_sorted.bam";
int length = 34;

/******************************************************************************
 *									GPU
 ******************************************************************************/

__device__ long long stringToUINT64GPU( char *s) 
{

	long long temp = 0;

	for( int i = 0; i < 17; i++){
		temp+= (s[i] == 'A') ? 1 << (3*i) : 0;
		temp+= (s[i] == 'C') ? 2 << (3*i) : 0;
		temp+= (s[i] == 'G') ? 3 << (3*i) : 0;
		temp+= (s[i] == 'T') ? 4 << (3*i) : 0;
		temp+= (s[i] == '-') ? 7 << (3*i) : 0;
	}

	return temp;
}

__global__ void convert_kernel(Read *bam_data)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	// Read in a single read from global memory
	Read ba = bam_data[idx];

	char left[17];
	char right[17];

	for(int i =0; i<17; i++){
		left[i]  = ba.sequence[i];
		right[i] = ba.sequence[i+17];
	}

	ba.left_sequence_half  = stringToUINT64GPU(left);
	ba.right_sequence_half = stringToUINT64GPU(right);

	// Save the read to global memory.
	bam_data[idx] = ba;

}


/******************************************************************************
 *									CPU
 ******************************************************************************/

int check_output( Read *gpu, int n)
{

	BamReader *br = new BamReader();
	br->Open(file);
	BamAlignment ba;
	ba.Position = -1;

	for (int i = 0; i < n; i++){
		while(ba.Position < 0)
			br->GetNextAlignment(ba);
		int offset    = (ba.IsReverseStrand()) ? ba.AlignedBases.length() - length : 0 ;
		string word   = ba.AlignedBases.substr(offset, length);
		Read bam = buildRead(word, length);

		if (  bam.left_sequence_half  != gpu[i].left_sequence_half || 
				bam.right_sequence_half  != gpu[i].right_sequence_half)
		{
			cout << "Error : "<< i << endl;
			cout << "GPU left = " << gpu[i].left_sequence_half << " | GPU Right = " << gpu[i].right_sequence_half << endl;
			cout << "CPU left = " << bam.left_sequence_half << " | CPU Right = " << bam.right_sequence_half << endl;
			return 0;
		}
	}

	return 1;
}

int main (int argc, char **argv) {
	
	// ------------------------------------------------------------------------
	// Read the BAM file
	// We're going to do a simple map/reduce on this data to prep data for GPU.
	// ------------------------------------------------------------------------
	int unique_reads = readFile(file, length, convert);
	
	/****************************************************************************
	 * GPU Setup 
	 ***************************************************************************/
	
	const long alignmentBytes = unique_reads * sizeof(Read);
	const long aBytes 	  = unique_reads * sizeof(BamAlignment);
	
	// --- Create & configure CUDA streams
	const int nStreams = 4;
	const int streamBytes = alignmentBytes / nStreams;

	// allocate CPU memory
	Read *a = 0;
	BamAlignment *alignments = 0;
	Read d_alignments[nStreams];
	float gpu_time = 0.0f;	// total time on GPU
	unsigned long int counter2=0;

	bool bFinalResults;
	// ------------------------------------------------------------------------
	// GPU Initialization
	// ------------------------------------------------------------------------
	int devID;
	hipDeviceProp_t deviceProps;
	StopWatchInterface *timer = NULL;

	// This will pick the best possible CUDA capable device
	devID = findCudaDevice(argc, (const char **)argv);

	// get device name
	checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
	printf("I'm using the CUDA device [%s]\n", deviceProps.name);
	
	// set kernel launch configuration
	dim3 threads = dim3(1024, 1);
	dim3 blocks  = dim3(unique_reads / threads.x, 1);

	// --- Choose which GPU to run on, change this on a multi-GPU system.
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(devID);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}


	// ------------------------------------------------------------------------
	// Memory Allocation
	// ------------------------------------------------------------------------

	checkCudaErrors(hipHostMalloc((void **)&a, alignmentBytes));
	memset(a, 0, alignmentBytes);

	checkCudaErrors(hipHostMalloc((void **)&alignments, aBytes));
	memset(alignments, 0, aBytes);

	// allocate GPU memory
	for (int i = 0; i < nStreams; ++i){
		checkCudaErrors( hipMalloc((Read **)&d_alignments[i], streamBytes));
		checkCudaErrors(hipMemset(&d_alignments[i], 0, streamBytes));
	}

	// ------------------------------------------------------------------------
	// Setup Streams & Timers
	// ------------------------------------------------------------------------

	// --- create cuda timers
	hipEvent_t start, stop;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));
	
	hipEvent_t timers[nStreams][4];
	for (int i = 0; i < nStreams; ++i){
		checkCudaErrors(hipEventCreate(&timers[i][0]));
		checkCudaErrors(hipEventCreate(&timers[i][1]));
		checkCudaErrors(hipEventCreate(&timers[i][2]));
		checkCudaErrors(hipEventCreate(&timers[i][3]));
	}

	sdkCreateTimer(&timer);
	sdkResetTimer(&timer);

	
	hipStream_t stream[nStreams];
	for (int i = 0; i < nStreams; ++i)
		checkCudaErrors( hipStreamCreate(&stream[i]) );

	checkCudaErrors(hipDeviceSynchronize());

	// ------------------------------------------------------------------------
	// Convert BAM Alignments to binary representations
	// ------------------------------------------------------------------------

	sdkStartTimer(&timer);
	hipEventRecord(start, 0);

	for(int i = 0 ; i < nStreams; i++){
	// hipEventRecord(time1, 0);
	// hipEventRecord(time2, 0);
		hipMemcpyAsync(d_alignments, a, alignmentBytes, hipMemcpyHostToDevice, stream[i]);
	//hipEventRecord(time3, 0);
		convert_kernel<<< 1,1 , 0, stream[i]>>>(d_alignments);
	// hipEventRecord(time4, 0);
		hipMemcpyAsync(a, d_alignments, alignmentBytes, hipMemcpyDeviceToHost, stream[i]);
	}

	hipEventRecord(stop, 0);
	sdkStopTimer(&timer);

	// have CPU do some work while waiting for stage 1 to finish
	while ( hipEventQuery(stop) == hipErrorNotReady)
		counter2++;

	checkCudaErrors(hipEventElapsedTime(&gpu_time, start, stop));

	// print the cpu and gpu times
	printf("time spent executing by the GPU: %.2f (ms) \n", gpu_time);
	printf("time spent by CPU in CUDA calls: %.2f (ms) \n", sdkGetTimerValue(&timer));
	printf("CPU executed %lu iterations while waiting for GPU to finish\n", counter2);

	// ------------------------------------------------------------------------
	// Check Correctness. 
	// ------------------------------------------------------------------------
	bFinalResults = (bool) check_output(a, unique_reads);

	// ------------------------------------------------------------------------
	// Verify the reads
	// ------------------------------------------------------------------------

	// ------------------------------------------------------------------------
	// Print out the new calls
	// ------------------------------------------------------------------------


	// ------------------------------------------------------------------------
	// End. 
	// ------------------------------------------------------------------------
	// release resources
Error:
	for (int i = 0; i < nStreams; ++i){
		checkCudaErrors(hipEventDestroy(timers[i][0]));
		checkCudaErrors(hipEventDestroy(timers[i][1]));
		checkCudaErrors(hipEventDestroy(timers[i][2]));
		checkCudaErrors(hipEventDestroy(timers[i][3]));
		checkCudaErrors( hipStreamDestroy(stream[i]) );
		checkCudaErrors(hipFree(&d_alignments[i] ));
	}
	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(stop));
	checkCudaErrors(hipHostFree(a));
	hipDeviceReset();

	exit(bFinalResults ? EXIT_SUCCESS : EXIT_FAILURE);
}

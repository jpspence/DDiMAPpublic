#include "hip/hip_runtime.h"
//============================================================================
// Name        : DDiMAP.cpp
// Author      : Androwis Abumoussa
// Version     :
// Copyright   : All Rights Reserved
// Description : DDiMAP in C
//============================================================================
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include "DDiMAP-lib.h"
#include <api/BamAlignment.h>
#include <api/BamReader.h>
#include <getopt.h>
#include <time.h>
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <string>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

using namespace BamTools;
using namespace std;

// Default file.
// char *file = "data/Burack_128F/128F_Gen1_Frag_WithBcl2Sanger_sorted.bam";
string file  = "data/128test_Gen1_example_sorted.bam";

/******************************************************************************
 *									GPU
 ******************************************************************************/

__device__ long long stringToUINT64GPU( char *s) 
{

	long long a = 1;
	long long c = 2;
	long long g = 3;
	long long t = 4;
	long long dash = 7;

	long long temp = 0;
	for( int i = 0; i < 17; i++){
		temp+= (s[i] == 'A') ? a 	<< (3*i) : 0;
		temp+= (s[i] == 'C') ? c 	<< (3*i) : 0;
		temp+= (s[i] == 'G') ? g 	<< (3*i) : 0;
		temp+= (s[i] == 'T') ? t 	<< (3*i) : 0;
		temp+= (s[i] == '-') ? dash << (3*i) : 0;
	}

	return temp;
}

__global__ void convert_kernel(Read *bam_data, int offset)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x + offset;

	// Read in a single read from global memory
	Read ba = bam_data[idx];

	char left[17];
	char right[17];

	for(int i =0; i<17; i++){
		left[i]  = ba.sequence[i];
		right[i] = ba.sequence[i+17];
	}

	ba.left_sequence_half  = stringToUINT64GPU(left);
	ba.right_sequence_half = stringToUINT64GPU(right);

	// Save the read to global memory.
	bam_data[idx] = ba;

}


/******************************************************************************
 *									CPU
 ******************************************************************************/

long n = 1024 * 1024 ;

int correct_output( Read *gpu)
{
	int length = 34;

	BamReader *br = new BamReader();
	br->Open(file);
	BamAlignment ba;
	ba.Position = -1;

	for (int i = 0; i < n; i++){
		while(ba.Position < 0)
			br->GetNextAlignment(ba);
		int offset    = (ba.IsReverseStrand()) ? ba.AlignedBases.length() - length : 0 ;
		string word   = ba.AlignedBases.substr(offset, length);
		Read bam = buildRead(word);

		if (  bam.left_sequence_half  != gpu[i].left_sequence_half || 
				bam.right_sequence_half  != gpu[i].right_sequence_half)
		{
			cout << "Error : "<< i << endl;
			cout << "GPU left = " << gpu[i].left_sequence_half << " | GPU Right = " << gpu[i].right_sequence_half << endl;
			cout << "CPU left = " << bam.left_sequence_half << " | CPU Right = " << bam.right_sequence_half << endl;
			return 0;
		}
	}

	return 1;
}

int main (int argc, char **argv) {


	// ------------------------------------------------------------------------
	// Parameters
	// ------------------------------------------------------------------------
	int c;

	static struct option long_options[] = {
			{"file", 	0, 0, 'f'},
			{NULL, 		0, NULL, 0}
	};

	int option_index = 0;
	while ((c = getopt_long(argc, argv, "f:", long_options, &option_index)) != -1) {

		switch (c) {
		case 'f':
			printf ("Parsing file :  %s \n",optarg);
			file = optarg;
			break;
		default:
			printf ("?? getopt returned character code 0%o ??\n", c);
		}
	}
	if (optind < argc) {
		printf ("non-option ARGV-elements: ");
		while (optind < argc)
			printf ("%s ", argv[optind++]);
		printf ("\n");
	}



	// ------------------------------------------------------------------------
	// Setup
	// ------------------------------------------------------------------------

	int devID;
	hipDeviceProp_t deviceProps;

	printf("[%s] - Starting...\n", argv[0]);

	// This will pick the best possible CUDA capable device
	devID = findCudaDevice(argc, (const char **)argv);

	// get device name
	checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
	printf("CUDA device [%s]\n", deviceProps.name);

	long alignmentBytes = n * sizeof(Read);
	long aBytes = n * sizeof(BamAlignment);

	// allocate host memory
	Read *a = 0;
	checkCudaErrors(hipHostMalloc((void **)&a, alignmentBytes));
	memset(a, 0, alignmentBytes);

	BamAlignment *alignments = 0;
	checkCudaErrors(hipHostMalloc((void **)&alignments, aBytes));
	memset(alignments, 0, aBytes);

	// allocate device memory
	Read *d_alignments=0;
	checkCudaErrors(hipMalloc((void **)&d_alignments, alignmentBytes));
	checkCudaErrors(hipMemset(d_alignments, 0, alignmentBytes));

	// set kernel launch configuration
	dim3 threads = dim3(512, 1);
	dim3 blocks  = dim3(n / threads.x, 1);

	// create cuda event handles
	hipEvent_t start, stop;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	StopWatchInterface *timer = NULL;
	sdkCreateTimer(&timer);
	sdkResetTimer(&timer);
	checkCudaErrors(hipDeviceSynchronize());

	float gpu_time = 0.0f;

	// ------------------------------------------------------------------------
	// ASYNC DDiMAP Kernel Execution
	// ------------------------------------------------------------------------

	// Read the bamfile
	BamReader *br = new BamReader();
	br->Open(file);
	BamAlignment ba;
	ba.Position = -1;
	int counter = 0;
	while(counter < n ){
		while(ba.Position < 0)
			br->GetNextAlignment(ba);
		a[counter] = convert(ba);
		counter++;
	}
	br->Close();

	hipStream_t stream0;
	hipStream_t stream1;
	hipStream_t stream2;
	hipStream_t stream3;

	// asynchronously issue work to the GPU (all to stream 0)
	sdkStartTimer(&timer);
	hipEventRecord(start, stream0);

	checkCudaErrors( hipStreamCreate(&stream0));
	checkCudaErrors( hipStreamCreate(&stream1));
	checkCudaErrors( hipStreamCreate(&stream2));
	checkCudaErrors( hipStreamCreate(&stream3));


	hipMemcpyAsync(&d_a[offset], &a[offset], streamBytes, stream[i]);
	kernel<<>>(d_a, offset);
	hipMemcpyAsync(&a[offset], &d_a[offset], streamBytes, stream[i]);

	hipMemcpyAsync(d_alignments, a, alignmentBytes/4, hipMemcpyHostToDevice, stream0);
	convert_kernel<<<blocks, threads, 0, stream0>>>(d_alignments, 0);
	hipMemcpyAsync(a, d_alignments, alignmentBytes/4, hipMemcpyDeviceToHost, stream0);

	int offset = n/4;
	hipMemcpyAsync(d_alignments[offset], a[offset], alignmentBytes/4, hipMemcpyHostToDevice, stream1);
	convert_kernel<<<blocks, threads, 0, stream1>>>(d_alignments, offset);
	hipMemcpyAsync(a[offset], d_alignments[offset], alignmentBytes/4, hipMemcpyDeviceToHost, stream1);

	offset += n/4;
	hipMemcpyAsync(d_alignments[offset], a[offset], alignmentBytes/4, hipMemcpyHostToDevice, stream2);
	convert_kernel<<<blocks, threads, 0, stream2>>>(d_alignments, offset);
	hipMemcpyAsync(a[offset], d_alignments[offset], alignmentBytes/4, hipMemcpyDeviceToHost, stream2);
	
	offset += n/4;
	hipMemcpyAsync(d_alignments[offset], a[offset], alignmentBytes/4, hipMemcpyHostToDevice, stream3);
	convert_kernel<<<blocks, threads, 0, stream3>>>(d_alignments, offset);
	hipMemcpyAsync(a[offset], d_alignments[offset], alignmentBytes/4, hipMemcpyDeviceToHost, stream3);


	hipEventRecord(stop, stream0);
	hipEventRecord(stop, stream1);
	hipEventRecord(stop, stream2);
	hipEventRecord(stop, stream3);
	sdkStopTimer(&timer);


	// have CPU do some work while waiting for stage 1 to finish
	unsigned long int counter2=0;
	while ( hipStreamQuery(stream0) == hipErrorNotReady ||
			hipStreamQuery(stream1) == hipErrorNotReady ||
			hipStreamQuery(stream2) == hipErrorNotReady ||
			hipStreamQuery(stream3) == hipErrorNotReady 
	)
	{
		counter2++;
	}
	checkCudaErrors(hipEventElapsedTime(&gpu_time, start, stop));

	// ------------------------------------------------------------------------
	// Check Correctness. 
	// ------------------------------------------------------------------------

	// print the cpu and gpu times
	printf("time spent executing by the GPU: %.2f (ms) \n", gpu_time);
	printf("time spent by CPU in CUDA calls: %.2f (ms) \n", sdkGetTimerValue(&timer));
	printf("CPU executed %lu iterations while waiting for GPU to finish\n", counter2);

	// check the output for correctness
	bool bFinalResults = (bool) correct_output(a);


	// ------------------------------------------------------------------------
	// End. 
	// ------------------------------------------------------------------------
	// release resources

	checkCudaErrors( hipStreamDestroy(stream0));
	checkCudaErrors( hipStreamDestroy(stream1));
	checkCudaErrors( hipStreamDestroy(stream2));
	checkCudaErrors( hipStreamDestroy(stream3));

	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(stop));
	checkCudaErrors(hipHostFree(a));
	checkCudaErrors(hipFree(d_alignments));

	hipDeviceReset();

	exit(bFinalResults ? EXIT_SUCCESS : EXIT_FAILURE);
}
